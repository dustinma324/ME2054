#include "hip/hip_runtime.h"
/*
 * Purpose: Demonstrate matrix multiplication in
 * CPU and GPU with global memory and shared memory usage
 * Date and time: 04/09/2014
 *
 * Last modified: Dustin (Ting-Hsuan) Ma
 * Date : November 20, 2018
 * Author: Inanc Senocak
 *
 * to compile blas: nvcc -lcublas -O2 gpu_matrixMultiply.cu -o GPU.exe
 * to execute: ./matrixMult.exe <m> <n> <k>
 */

#include "hipblas.h"
#include "timer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <time.h>

#define BLOCKSIZE 16

typedef double REAL;
typedef int    INT;

void printMatrix(REAL *matrix, const int nrow, const int ncol)
{
	int i, j, idx;

	for (j = 0; j < nrow; j++) {
		for (i = 0; i < ncol; i++) {
			idx = i + j * ncol;
			printf("%8.2f ; ", matrix[idx]);
		}
		printf("\n");
	}
	printf("\n");
}

void InitializeMatrices(REAL *a, REAL *b, const int M, const int N, const int K)
{
	int i, j, idx;

	// initialize matrices a & b
	for (j = 0; j < M; j++) {
		for (i = 0; i < N; i++) {
			idx    = i + j * N;
			a[idx] = (REAL) idx;
		}
	}

	for (j = 0; j < N; j++) {
		for (i = 0; i < K; i++) {
			idx    = i + j * K;
			b[idx] = (REAL) idx;
		}
	}
}

__global__ void matrixMultiplyGPU_gl(REAL *a, REAL *b, REAL *c, const int M, const int N,
                                     const int K)
{
	// Block index

	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Row index of matrices a and c

	int row = by * BLOCKSIZE + ty;

	// Column index of matrices a and b
	int col = bx * BLOCKSIZE + tx;

	REAL C_temp = 0.;

	if (row < M && col < K){
		for (int k = 0; k < N; k++)
			C_temp += a[k + row * N] * b[col + k * K];

	c[col + row * K] = C_temp;
	}
}

int main(INT argc, char *argv[])
{
	if (argc < 3) {
		perror("Command-line usage: executableName <M> <N> <K>");
		exit(1);
	}

	int M = atof(argv[1]);
	int N = atof(argv[2]);
	int K = atof(argv[3]);

	REAL *a_d, *b_d, *c_d, *d_d, *e_d;

	hipMallocManaged(&a_d, M * N * sizeof(*a_d));
	hipMallocManaged(&b_d, N * K * sizeof(*b_d));
	hipMallocManaged(&c_d, M * K * sizeof(*c_d)); // Used for GPU
	hipMallocManaged(&d_d, M * K * sizeof(*d_d)); // Used for cublasDDOT
	hipMallocManaged(&e_d, M * K * sizeof(*e_d)); // Used for cublasDAXPY

	InitializeMatrices(a_d, b_d, M, N, K);

	// Setting up GPU enviorment
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid((K + 15) / BLOCKSIZE, (M + 15) / BLOCKSIZE);

	float elapsedTime_gpu, elapsedTime_DDOT, elapsedTime_DAXPY;

	printf("=====MultKernel=====\n");
	hipEvent_t timeStart, timeStop; // WARNING!!! use events only to time the device
	hipEventCreate(&timeStart);
	hipEventCreate(&timeStop);
	hipEventRecord(timeStart, 0);

	matrixMultiplyGPU_gl<<<dimGrid, dimBlock>>>(a_d, b_d, c_d, M, N, K);

	hipDeviceSynchronize();
	hipEventRecord(timeStop, 0);
	hipEventSynchronize(timeStop);
	hipEventElapsedTime(&elapsedTime_gpu, timeStart, timeStop);

	printMatrix( c_d, M, K );
	//printf("C[2] = %3.1f\n", c_d[2]);
	printf("elapsed wall time (GPU) = %5.2f ms\n", elapsedTime_gpu);

	printf("=====cublasDDOT=====\n");
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipEventRecord(timeStart, 0);

	for (int i = 0; i < M; i++) {
		for (int j = 0; j < K; j++) {
			hipblasDdot(handle, N, a_d + j * N, 1, b_d + i, K, d_d + i + j * K);
		}
	}

	hipEventRecord(timeStop, 0);
	hipEventSynchronize(timeStop);
	hipEventElapsedTime(&elapsedTime_DDOT, timeStart, timeStop);
	printMatrix( d_d, M, K );
	//printf("D[2] = %3.1f\n", d_d[2]);
	printf("elapsed wall time (cublasDDOT) = %5.2f ms\n", elapsedTime_DDOT);

	printf("=====cublasDAXPY=====\n");
	hipEventRecord(timeStart, 0);

	for (int j = 0; j < M; j++) {
		for (int i = 0; i < K; i++) {
			hipblasDaxpy(handle, M, b_d + j + i * K, a_d + i, N, e_d + j, K);
		}
	}

	hipEventRecord(timeStop, 0);
	hipEventSynchronize(timeStop);
	hipEventElapsedTime(&elapsedTime_DAXPY, timeStart, timeStop);
	printMatrix( e_d, M, K );
	//printf("E[2] = %3.1f\n", e_d[2]);
	printf("elapsed wall time (cublasDAXPY) = %5.2f ms\n", elapsedTime_DAXPY);
	printf("\n");
	hipblasDestroy(handle);

	// Deallocating Memory
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(e_d);
	hipEventDestroy(timeStart);
	hipEventDestroy(timeStop);

	return (EXIT_SUCCESS);
}
