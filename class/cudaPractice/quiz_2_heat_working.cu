#include "hip/hip_runtime.h"
/***********************************************************************
 * QUIZ INSTRUCTIONS
 *
 * you are given a partially completed code. Read the code line by line
 * and implement the necessary steps to compute the 1d heat conduction
 * problem given in the handout.
 *
 ***********************************************************************
 *
 * Numerical and analytical solution of the 1D heat conduction problem
 *
 * Author: enter your name
 * Date: enter today's date
 *
 * gcc -O2 -lm -std=c99 1d_heat.c -o heat_1d.exe
 *
 * to execute: ./heat_1d.exe <simulation end time (seconds)>
 *
 * nvcc -O2 quiz_2_heat_working.cu -DSINGLE=1 -o run.exe
 */

#include "timer.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>

#define LX 3.0f /* length of the domain in x-direction  */
#define NX 32   /* includes boundary points on both end */
#define DX LX / ((REAL)(NX - 1))
#define ALPHA 1.0f
#define DT 0.25f * DX *DX / ALPHA
#define BLOCK_SIZE 256

#define RESTRICT __restrict__

#ifndef SINGLE
typedef double REAL;
typedef int    INT;
#define PI M_PI
#else
typedef float REAL;
typedef int   INT;
#define PI M_PI
#endif

__global__ void solveHeat_1D (REAL *RESTRICT unew, const REAL *RESTRICT u, const REAL *RESTRICT x)
{
    INT  i = threadIdx.x + blockIdx.x * blockDim.x;
    REAL dxi = 1.f / (DX * DX);
    REAL xc, source;

    if ( i > 0 && i < NX-1 ){
        xc     = x[ i ];
        source = -(xc * xc - 4.f * xc + 2.f) * exp(-xc); // source term
        unew[ i ] = (ALPHA * (u[ i + 1 ] - 2.0f * u[ i ] + u[ i - 1 ]) * dxi + source) * DT + u[ i ];
    }
}
void exactSolution(REAL *RESTRICT uExact, const REAL *RESTRICT x)
{
    INT i;
    for (i = 0; i < NX; i++) {
        uExact[ i ] = x[ i ] * x[ i ] * exp(-x[ i ]);
    }
}

void meshGrid(REAL *RESTRICT x)
{
    INT i;
    for (i = 0; i < NX; i++) {
        x[ i ] = DX * (( REAL ) i);
    }
}

void writeOutput(const REAL *RESTRICT x, const REAL *RESTRICT uExact, const REAL *RESTRICT u)
{
    INT   i;
    FILE *output;
    output = fopen("1d_heat.dat", "w");

    for (i = 0; i < NX; i++) {
        fprintf(output, "%10f %10f %10f\n", x[ i ], uExact[ i ], u[ i ]);
    }
    fclose(output);
}

INT main(INT argc, char *argv[])
{
    if (argc < 2) {
        perror("Command-line usage: executableName <end Time (seconds)>");
        exit(1);
    }

    REAL endTime = atof(argv[ 1 ]);

    REAL *uExact, *x;
    REAL *unew, *u, *tmp;

    //  allocate heap memory here for arrays needed in the solution algorithm
    //  read the code carefully to determine those variables

    hipMallocManaged(&unew,NX* sizeof(*unew));
    hipMallocManaged(&u,NX* sizeof(*u));
    hipMallocManaged(&x,NX* sizeof(*x));

    uExact =(REAL*)calloc(NX, sizeof(*uExact));

    // calculate the x coordinates of each computational point
    meshGrid(x);
    // compute the exact solution to the 1D heat conduction problem
    exactSolution(uExact, x);

    // apply boundary conditions (make sure to apply boundary conditions to both u and unew)
    u[ 0 ]         = 0.f;
    unew[ 0 ]      = 0.f;
    unew[ NX - 1 ] = uExact[ NX - 1 ];
    u[ NX - 1 ]    = uExact[ NX - 1 ];

    REAL time = 0.f;

    int nBlocks = (NX + BLOCK_SIZE - 1) / BLOCK_SIZE;
    REAL   elapsedTime;   // in float because it is recorded in ms

    hipEvent_t timeStart, timeStop; // hipEvent_t initializes variable used in event time
    hipEventCreate(&timeStart);
    hipEventCreate(&timeStop);
    hipEventRecord(timeStart, 0);

    while (time < endTime) {
        // call the solveHeat_1D( ) function here with correct parameters
        // and necessary updates on the solution array
        solveHeat_1D<<<nBlocks,BLOCK_SIZE>>>(unew, u, x);
	hipDeviceSynchronize();
	// swap pointers
        tmp  = unew;
        unew = u;
        u    = tmp;
	// incrementing the time
        time += DT;
    }

    hipEventRecord(timeStop, 0);
    hipEventSynchronize(timeStop);
    hipEventElapsedTime(&elapsedTime, timeStart, timeStop);


    // call the writeOutput( ) function here with correct parameters

    writeOutput(x, uExact, u);

    hipFree(unew);
    hipFree(u);
    free(uExact);
    hipFree(x);

    return EXIT_SUCCESS;
}
